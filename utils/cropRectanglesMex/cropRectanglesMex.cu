#include "hip/hip_runtime.h"

#include "mex.h"
#include "gpu/mxGPUArray.h"

#include <npp.h>

#include <math.h> 

#define MATLAB_ASSERT(expr,msg) if (!(expr)) { mexErrMsgTxt(msg);}

#if !defined(MX_API_VER) || MX_API_VER < 0x07030000
typedef size_t mwSize;
typedef size_t mwIndex;
#endif

void mexFunction(int nlhs, mxArray *plhs[], 
    int nrhs, const mxArray *prhs[])
{
	MATLAB_ASSERT( nrhs == 3, "cropRectanglesMex: Wrong number of input parameters: expected 3");
    MATLAB_ASSERT( nlhs == 1, "cropRectanglesMex: Wrong number of output arguments: expected 1");
	
	// Fix input parameter order:
	const mxArray *imInPtr = (nrhs >= 0) ? prhs[0] : NULL; // image
	const mxArray *bbInPtr = (nrhs >= 1) ? prhs[1] : NULL; // bounding boxes
	const mxArray *szInPtr = (nrhs >= 2) ? prhs[2] : NULL; // output image size
	
	// Fix output parameter order:
	mxArray **cropsOutPtr = (nlhs >= 1) ? &plhs[0] : NULL; // croped and resized patches
	
	// Get the image
	MATLAB_ASSERT(!mxIsGPUArray(imInPtr) && mxGetClassID(imInPtr) == mxSINGLE_CLASS, "cropRectanglesMex: the image should be of type SINGLE in RAM (not GPU)");
	MATLAB_ASSERT(mxGetNumberOfDimensions(imInPtr) == 3, "cropRectanglesMex: the image is not 3-dimensional");
	MATLAB_ASSERT(mxGetPi(imInPtr) == NULL, "cropRectanglesMex: image should not be complex");

    const mwSize* dimensions = mxGetDimensions(imInPtr);
	mwSize imageHeight = dimensions[0];
	mwSize imageWidth = dimensions[1];
	mwSize numChannels = dimensions[2];
	MATLAB_ASSERT(numChannels == 3, "cropRectanglesMex: image should contain 3 channels");

	float* imageData = (float*) mxGetData(imInPtr);

	// get bounding boxes
	MATLAB_ASSERT(!mxIsGPUArray(bbInPtr) && mxGetClassID(bbInPtr) == mxDOUBLE_CLASS, "cropRectanglesMex: <boundingBoxes> input is not of type double in RAM (not GPU)");
	MATLAB_ASSERT(mxGetNumberOfDimensions(bbInPtr) == 2, "cropRectanglesMex: <boundingBoxes> input is not 2-dimensional");
	MATLAB_ASSERT(mxGetPi(bbInPtr) == NULL, "cropRectanglesMex: <boundingBoxes> input should not be complex");
	MATLAB_ASSERT(mxGetN(bbInPtr) == 4, "cropRectanglesMex: <boundingBoxes> input should be of size #boundingBoxes x 4");
	
	mwSize numBb = mxGetM(bbInPtr);
	double* bbData = (double*) mxGetData(bbInPtr); // y1, x1, y2, x2

	// get output size
	MATLAB_ASSERT(!mxIsGPUArray(szInPtr) && mxGetClassID(szInPtr) == mxDOUBLE_CLASS, "cropRectanglesMex: <outputSize> input is not of type double in RAM (not GPU)");
	MATLAB_ASSERT(mxGetNumberOfElements(szInPtr) == 2, "cropRectanglesMex: <outputSize> input should contain 2 numbers");
	MATLAB_ASSERT(mxGetPi(szInPtr) == NULL, "cropRectanglesMex: <outputSize> input should not be complex");
	
	double* outputSizeData = (double*) mxGetData(szInPtr);
	int targetHeight = (int) (outputSizeData[0] + 0.5);
	int targetWidth = (int) (outputSizeData[1] + 0.5);

	// initialize GPU
	mxInitGPU();

	// copy image to the GPU
	mxGPUArray const *inputImage;
    float const *d_inputImage;
	inputImage = mxGPUCreateFromMxArray(imInPtr);
	d_inputImage = (float const *)(mxGPUGetDataReadOnly(inputImage));

	// allocate memory for the output
    mxGPUArray *outputData;
    float *d_outputData;
	const mwSize outputDimensions[4] = { targetHeight, targetWidth, numChannels, numBb };
	outputData = mxGPUCreateGPUArray(4, outputDimensions, mxSINGLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES) ; //MX_GPU_DO_NOT_INITIALIZE);
	d_outputData = (float *)(mxGPUGetData(outputData));

	// initialize some cropping arguments
	NppiSize nppiImageSize = {};
	nppiImageSize.width = imageHeight; // CAUTION: NPPI thinks that the image is transposed 
	nppiImageSize.height = imageWidth;

	int channelValueSize = sizeof(float);
	int imageStep = imageHeight * channelValueSize;
	int targetStep = targetHeight * channelValueSize;

    NppiRect targetRect = {};
    targetRect.x = 0;
    targetRect.y = 0;
    targetRect.width = targetHeight;
    targetRect.height = targetWidth;
	
	// the main loop over bounding boxes
	for(int iBb = 0; iBb < numBb; ++iBb) {

		double y1 = bbData[ iBb ] - 1;
		double x1 = bbData[ iBb + numBb ] - 1;
		double y2 = bbData[ iBb + numBb * 2 ] - 1;
		double x2 = bbData[ iBb + numBb * 3 ] - 1;

	    double nXFactor = double( targetHeight ) / ( y2 - y1 + 1 );
		double nYFactor = double( targetWidth ) / ( x2 - x1 + 1 );
		double nXShift = -nXFactor * (double(y1) + 0.5) + 0.5;
		double nYShift = -nYFactor * (double(x1) + 0.5) + 0.5;

		NppiRect sourceRect = {};
    	sourceRect.x = (int) floor(y1);
    	sourceRect.y = (int) floor(x1);
    	sourceRect.width =  (int) ceil(y2 - y1 + 1);
    	sourceRect.height = (int) ceil(x2 - x1 + 1);
    	if (sourceRect.width <= 1) {
    		sourceRect.width = 2;
    	}
    	if (sourceRect.height <= 1) {
    		sourceRect.height = 2;
    	}

    	// adjust bounding box bounds if it is outside of the image
    	if (sourceRect.x < 0) {
    		sourceRect.width = sourceRect.width + sourceRect.x;
    		sourceRect.x = 0.0;
    	}
    	if (sourceRect.y < 0) {
    		sourceRect.height = sourceRect.height + sourceRect.y;
    		sourceRect.y = 0.0;
    	}
    	if (sourceRect.width > imageHeight - sourceRect.x + 1) {
    		sourceRect.width = imageHeight - sourceRect.x + 1;
    	}
    	if (sourceRect.height > imageWidth - sourceRect.y + 1) {
    		sourceRect.height = imageWidth - sourceRect.y + 1;
    	}

    	float *curOutput = d_outputData + numChannels * targetHeight * targetWidth * iBb;
		const float *pSrc[3] = { d_inputImage, d_inputImage + imageHeight * imageWidth, d_inputImage + 2 * imageHeight * imageWidth};
		float *pDst[3] = { curOutput, curOutput + targetHeight * targetWidth, curOutput + 2 * targetHeight * targetWidth};

        // When NPP_CHECK_NPP catches an error it throws an exception
        // If the exception is not caught, we can get a memory leak on a GPU
        try{
             NppStatus exitCode = nppiResizeSqrPixel_32f_P3R (
        		pSrc,  // const Npp32f *pSrc, 
            	nppiImageSize, // nppiSize oSrcSize, 
                imageStep, // int nSrcStep, 
                sourceRect, // NppiRect oSrcROI, 
                pDst, // Npp8u *pDst, 
                targetStep, // int nDstStep, 
                targetRect, // NppiRect oDstROI, 
                nXFactor, nYFactor, nXShift, nYShift, 
                NPPI_INTER_CUBIC //int eInterpolation
                );
            if (exitCode != NPP_SUCCESS) {
                mexPrintf("nppiResizeSqrPixel_32f_P3R returns exit code %d, see http://cseweb.ucsd.edu/classes/wi15/cse262-a/static/cuda-5.5-doc/pdf/NPP_Library.pdf for the description of exit code.\n", exitCode);
                MATLAB_ASSERT(exitCode == NPP_SUCCESS, "cropRectanglesMex: nppiResizeSqrPixel_32f_P3R returns bad exit code");
            }
        } catch (...) {
            // free GPU memory
            mxGPUDestroyGPUArray(outputData);
            mxGPUDestroyGPUArray(inputImage);
            throw;
        }

    }
	
	*cropsOutPtr = mxGPUCreateMxArrayOnGPU(outputData);

	// do not forget to free GPU memory
	mxGPUDestroyGPUArray(outputData);
	mxGPUDestroyGPUArray(inputImage);
}
